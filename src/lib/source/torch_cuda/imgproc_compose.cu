#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/torch/imgproc.h"
#include "../generic/imageproc.hxx"
#include "grid.cu.hxx"

#ifdef __cuda_cuda_h__

template <typename T>
ABCDK_INVOKE_GLOBAL void _abcdk_torch_imgproc_compose_2d2d_cuda(int channels, bool packed,
                                                                T *panorama, size_t panorama_w, size_t panorama_ws, size_t panorama_h,
                                                                T *compose, size_t compose_w, size_t compose_ws, size_t compose_h,
                                                                uint32_t *scalar, size_t overlap_x, size_t overlap_y, size_t overlap_w, bool optimize_seam)
{
    size_t tid = abcdk::cuda::grid::get_tid(2, 2);

    abcdk::generic::imageproc::compose<T>(channels, packed,
                                          panorama, panorama_w, panorama_ws, panorama_h,
                                          compose, compose_w, compose_ws, compose_h,
                                          scalar, overlap_x, overlap_y, overlap_w, optimize_seam, tid);
}

template <typename T>
ABCDK_INVOKE_HOST int _abcdk_torch_imgproc_compose_cuda(int channels, bool packed,
                                                        T *panorama, size_t panorama_w, size_t panorama_ws, size_t panorama_h,
                                                        T *compose, size_t compose_w, size_t compose_ws, size_t compose_h,
                                                        uint32_t *scalar, size_t overlap_x, size_t overlap_y, size_t overlap_w, bool optimize_seam)
{
    void *gpu_scalar;
    uint3 dim[2];

    assert(panorama != NULL && panorama_w > 0 && panorama_ws > 0 && panorama_h > 0);
    assert(compose != NULL && compose_w > 0 && compose_ws > 0 && compose_h > 0);
    assert(scalar != NULL); // && overlap_x >= 0 && overlap_y >= 0 && overlap_w >= 0);

    gpu_scalar = abcdk_torch_copyfrom_cuda(scalar, channels * sizeof(uint32_t), 1);
    if (!gpu_scalar)
        return -1;

    /*2D-2D*/
    abcdk::cuda::grid::make_dim_dim(dim, compose_w * compose_h, 64);

    _abcdk_torch_imgproc_compose_2d2d_cuda<T><<<dim[0], dim[1]>>>(channels, packed,
                                                                  panorama, panorama_w, panorama_ws, panorama_h,
                                                                  compose, compose_w, compose_ws, compose_h,
                                                                  (uint32_t *)gpu_scalar, overlap_x, overlap_y, overlap_w, optimize_seam);
    abcdk_torch_free_cuda(&gpu_scalar);

    return 0;
}

__BEGIN_DECLS

int abcdk_torch_imgproc_compose_cuda(abcdk_torch_image_t *panorama, abcdk_torch_image_t *compose,
                                     uint32_t scalar[], size_t overlap_x, size_t overlap_y, size_t overlap_w, int optimize_seam)
{
    int dst_depth;

    assert(panorama != NULL && compose != NULL && scalar != NULL);
    assert(panorama->pixfmt == compose->pixfmt);
    assert(panorama->pixfmt == ABCDK_TORCH_PIXFMT_GRAY8 ||
           panorama->pixfmt == ABCDK_TORCH_PIXFMT_RGB24 ||
           panorama->pixfmt == ABCDK_TORCH_PIXFMT_BGR24 ||
           panorama->pixfmt == ABCDK_TORCH_PIXFMT_RGB32 ||
           panorama->pixfmt == ABCDK_TORCH_PIXFMT_BGR32);

    dst_depth = abcdk_torch_pixfmt_channels(panorama->pixfmt);

    return _abcdk_torch_imgproc_compose_cuda<uint8_t>(dst_depth, true,
                                                      panorama->data[0], panorama->width, panorama->stride[0], panorama->height,
                                                      compose->data[0], compose->width, compose->stride[0], compose->height,
                                                      scalar, overlap_x, overlap_y, overlap_w, optimize_seam);
}

__END_DECLS

#else // __cuda_cuda_h__

__BEGIN_DECLS

int abcdk_torch_imgproc_compose_cuda(abcdk_torch_image_t *panorama, abcdk_torch_image_t *compose,
                                     uint32_t scalar[], size_t overlap_x, size_t overlap_y, size_t overlap_w, int optimize_seam)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}

__END_DECLS

#endif // __cuda_cuda_h__