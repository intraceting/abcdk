/*
 * This file is part of ABCDK.
 * 
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 * 
 */
#include "abcdk/nvidia/context.h"

__BEGIN_DECLS

#ifdef __cuda_cuda_h__

void abcdk_cuda_ctx_destroy(hipCtx_t *ctx)
{
    hipCtx_t ctx_p = NULL;

    ctx_p = *ctx;
    *ctx = NULL;

    hipCtxDestroy(ctx_p);
}

static pthread_once_t _abcdk_cuda_ctx_key_init_status = PTHREAD_ONCE_INIT;
static pthread_key_t _abcdk_cuda_ctx_key = 0xFFFFFFFF;

static void _abcdk_cuda_ctx_key_init()
{
    pthread_key_create(&_abcdk_cuda_ctx_key,NULL);
}

hipCtx_t abcdk_cuda_ctx_create(int device, int flag)
{    
    hipCtx_t ctx = NULL;
    hipDevice_t dev_ctx;
    hipError_t cu_chk;
    int chk;

    assert(device >= 0);

    /*初始化一次。*/
    chk = pthread_once(&_abcdk_cuda_ctx_key_init_status,_abcdk_cuda_ctx_key_init);
    assert(chk == 0);

    cu_chk = hipDeviceGet(&dev_ctx, device);
    if (cu_chk != hipSuccess)
        goto ERR;

    cu_chk = hipCtxCreate(&ctx, flag, dev_ctx);
    if (cu_chk != hipSuccess)
        goto ERR;

    return ctx;

ERR:

    abcdk_cuda_ctx_destroy(&ctx);

    return NULL;
}

int abcdk_cuda_ctx_push(hipCtx_t ctx)
{
    hipError_t cu_chk;

    assert(ctx != NULL);

    /*绑定到设备。*/
    cu_chk = hipCtxPushCurrent(ctx);
    if (cu_chk != hipSuccess)
        return -1;
    
    return 0;
}

hipCtx_t abcdk_cuda_ctx_pop()
{
    hipCtx_t old_ctx = NULL;

    /*解除设备绑定。*/
    hipCtxPopCurrent(&old_ctx);

    return old_ctx;
}

int abcdk_cuda_ctx_setspecific(hipCtx_t ctx)
{
    int chk;

    /*绑定到线程。*/
    chk = pthread_setspecific(_abcdk_cuda_ctx_key, ctx);
    if (chk != 0)
        return -1;
    
    return 0;
}

hipCtx_t abcdk_cuda_ctx_getspecific()
{
    hipCtx_t old_ctx = NULL;

    old_ctx = (hipCtx_t)pthread_getspecific(_abcdk_cuda_ctx_key);
    ABCDK_ASSERT(old_ctx != NULL, TT("当前线程尚未绑定CUDA环境。"));

    return old_ctx;
}



#else // __cuda_cuda_h__

void abcdk_cuda_ctxt_destroy(hipCtx_t *ctx)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return ;
}

hipCtx_t abcdk_cuda_ctxt_create(int device, int flag)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return NULL;
}

int abcdk_cuda_ctx_push(hipCtx_t ctx)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}

hipCtx_t abcdk_cuda_ctx_pop()
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return NULL;
}

int abcdk_cuda_ctx_setspecific()
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}

hipCtx_t abcdk_cuda_ctx_getspecific()
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return NULL;
}

#endif //__cuda_cuda_h__


__END_DECLS