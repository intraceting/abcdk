/*
 * This file is part of ABCDK.
 * 
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 * 
 */
#include "abcdk/nvidia/nvidia.h"

__BEGIN_DECLS

#ifdef __cuda_cuda_h__

int abcdk_cuda_init(uint32_t flags)
{
    hipError_t chk;
    chk = hipInit(flags);
    if(chk != hipSuccess)
        return -1;
    
    return 0;
}

int abcdk_cuda_get_runtime_version(int *minor)
{
    int num_ver = 0;
    int major = -1;
    hipError_t chk;

    chk = hipRuntimeGetVersion(&num_ver);
    if (chk != hipSuccess)
        return -1;
    
    major = num_ver / 1000;

    if (minor)
        *minor = (num_ver % 1000) / 10;

    return major;
}

int abcdk_cuda_get_device_name(char name[256], int device)
{
    struct hipDeviceProp_t prop;
    hipError_t chk;

    assert(name != NULL && device >= 0);

    chk = hipGetDeviceProperties(&prop, device);
    if (chk != hipSuccess)
        return -1;

    strncpy(name, prop.name, 256);

    return 0;
}

#else // __cuda_cuda_h__

int abcdk_cuda_ctx_init(uint32_t flags)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}

int abcdk_cuda_get_runtime_version(int *minor)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}

int abcdk_cuda_get_device_name(char name[256], int device)
{
    abcdk_trace_printf(LOG_WARNING, TT("当前环境在构建时未包含CUDA工具。"));
    return -1;
}


#endif //__cuda_cuda_h__


__END_DECLS