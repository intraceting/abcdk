#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/tensor.h"
#include "grid.cu.hxx"
#include "util.cu.hxx"

#ifdef __cuda_cuda_h__

template <class ST, class DT>
ABCDK_CUDA_GLOBAL void _abcdk_cuda_tensor_blob_2d2d(int channels, bool revert,
                                                    bool dst_packed, DT *dst, size_t dst_ws,
                                                    bool src_packed, ST *src, size_t src_ws,
                                                    size_t w, size_t h, float *scale, float *mean, float *std)
{
    size_t tid = abcdk::cuda::grid_get_tid(2, 2);

    size_t y = tid / w;
    size_t x = tid % w;

    if (x >= w || y >= h)
        return;

    for (size_t z = 0; z < channels; z++)
    {
        size_t src_of = abcdk::cuda::off<ST>(src_packed, w, src_ws, h, channels, 0, x, y, z);
        size_t dst_of = abcdk::cuda::off<DT>(dst_packed, w, dst_ws, h, channels, 0, x, y, z);

        ST *src_p = abcdk::cuda::ptr<ST>(src, src_of);
        DT *dst_p = abcdk::cuda::ptr<DT>(dst, dst_of);

        if (revert)
            *dst_p = (((DT)*src_p * std[z]) + mean[z]) * scale[z];
        else
            *dst_p = (((DT)*src_p / scale[z]) - mean[z]) / std[z];
    }
}

template <class ST, class DT>
ABCDK_CUDA_HOST int _abcdk_cuda_tensor_blob(int channels, bool revert,
                                            bool dst_packed, DT *dst, size_t dst_ws,
                                            bool src_packed, ST *src, size_t src_ws,
                                            size_t w, size_t h, float *scale, float *mean, float *std)
{
    void *gpu_scale = NULL, *gpu_mean = NULL, *gpu_std = NULL;
    uint3 dim[2];

    gpu_scale = abcdk_cuda_copyfrom(scale, channels * sizeof(float), 1);
    gpu_mean = abcdk_cuda_copyfrom(mean, channels * sizeof(float), 1);
    gpu_std = abcdk_cuda_copyfrom(std, channels * sizeof(float), 1);

    if (!gpu_scale || !gpu_mean || !gpu_std)
    {
        abcdk_cuda_free(&gpu_scale);
        abcdk_cuda_free(&gpu_mean);
        abcdk_cuda_free(&gpu_std);
        return -1;
    }

    /*2D-2D*/
    abcdk::cuda::grid_make_2d2d(dim, w * h, 64);

    _abcdk_cuda_tensor_blob_2d2d<ST, DT><<<dim[0], dim[1]>>>(channels, revert,
                                                             dst_packed, dst, dst_ws,
                                                             src_packed, src, src_ws,
                                                             w, h, (float *)gpu_scale, (float *)gpu_mean, (float *)gpu_std);

    abcdk_cuda_free(&gpu_scale);
    abcdk_cuda_free(&gpu_mean);
    abcdk_cuda_free(&gpu_std);
    return 0;
}

int abcdk_cuda_tensor_blob_8u_to_32f_3r(int dst_packed, float *dst, size_t dst_ws,
                                        int src_packed, uint8_t *src, size_t src_ws,
                                        size_t w, size_t h, float scale[3], float mean[3], float std[3])
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(scale != NULL && mean != NULL && std != NULL);

    return _abcdk_cuda_tensor_blob<uint8_t, float>(3, false, dst_packed, dst, dst_ws, src_packed, src, src_ws, w, h, scale, mean, std);
}

int abcdk_cuda_tensor_blob_32f_to_8u_3r(int dst_packed, uint8_t *dst, size_t dst_ws,
                                        int src_packed, float *src, size_t src_ws,
                                        size_t w, size_t h, float scale[3], float mean[3], float std[3])
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(scale != NULL && mean != NULL && std != NULL);

    return _abcdk_cuda_tensor_blob<float, uint8_t>(3, true, dst_packed, dst, dst_ws, src_packed, src, src_ws, w, h, scale, mean, std);
}

#endif // __cuda_cuda_h__
