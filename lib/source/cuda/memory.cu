/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2021 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/memory.h"
#include "grid.cu.hxx"

#ifdef HAVE_CUDA

void abcdk_cuda_free(void **data)
{
    void *data_p;

    if(!data || !*data)
        return;

    data_p = *data;
    *data = NULL;

    hipFree(data_p);
}


void *abcdk_cuda_alloc(size_t size)
{
    void *data;
    hipError_t chk;

    assert(size > 0);

    chk = hipMalloc(&data,size);
    if(chk != hipSuccess)
        return NULL;

    return data;
}

int abcdk_cuda_memcpy(void *dst, const void *src, size_t size, int dst_in_host, int src_in_host)
{
    hipMemcpyKind kind = hipMemcpyDefault;
    hipError_t chk;

    assert(dst != NULL && src != NULL && size > 0);

    if (src_in_host && dst_in_host)
        kind = hipMemcpyHostToHost;
    else if (src_in_host)
        kind = hipMemcpyHostToDevice;
    else if (dst_in_host)
        kind = hipMemcpyDeviceToHost;
    else
        kind = hipMemcpyDeviceToDevice;

    chk = hipMemcpy(dst, src, size, kind);

    if (chk != hipSuccess)
        return -1;

    return 0;
}

int abcdk_cuda_memcpy_2D(void *dst, size_t dst_pitch, size_t dst_x_bytes, size_t dst_y,
                         const void *src, size_t src_pitch, size_t src_x_bytes, size_t src_y,
                         size_t roi_width_bytes, size_t roi_height, int dst_in_host, int src_in_host)
{
    hip_Memcpy2D copy_args = {0};
    hipError_t chk;

    assert(dst != NULL && src != NULL && roi_width_bytes > 0 && roi_height > 0);

    copy_args.dstXInBytes = dst_x_bytes;
    copy_args.dstY = dst_y;
    copy_args.dstMemoryType = (dst_in_host ? hipMemoryTypeHost : hipMemoryTypeDevice);
    copy_args.dstHost = (dst_in_host ? dst : NULL);
    copy_args.dstDevice = (hipDeviceptr_t)(dst_in_host ? NULL : dst);
    copy_args.dstPitch = dst_pitch;
    
    copy_args.srcXInBytes = src_x_bytes;
    copy_args.srcY = src_y;
    copy_args.srcMemoryType = (src_in_host ? hipMemoryTypeHost : hipMemoryTypeDevice);
    copy_args.srcHost = (src_in_host ? src : NULL);
    copy_args.srcDevice = (hipDeviceptr_t)(src_in_host ? NULL : src);
    copy_args.srcPitch = src_pitch;
    
    copy_args.WidthInBytes = roi_width_bytes;
    copy_args.Height = roi_height;
    
    chk = hipMemcpyParam2D(&copy_args);

    if (chk != hipSuccess)
        return -1;

    return 0;
}

void *abcdk_cuda_copyfrom(const void *src,size_t size,int src_in_host)
{
    void *dst;
    int chk;

    assert(src != NULL && size > 0);

    dst = abcdk_cuda_alloc(size);
    if(!dst)
        return NULL;

    chk = abcdk_cuda_memcpy(dst,src,size,0,src_in_host);
    if(chk == 0)
        return dst;

    abcdk_cuda_free(&dst);
    return NULL;
}

#endif //HAVE_CUDA
