#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/memory.h"
#include "../impl/invoke.hxx"
#include "grid.cu.hxx"

#ifdef __cuda_cuda_h__

void abcdk_cuda_free(void **data)
{
    void *data_p;

    if (!data || !*data)
        return;

    data_p = *data;
    *data = NULL;

    hipFree(data_p);
}

void *abcdk_cuda_alloc(size_t size)
{
    void *data;
    hipError_t chk;

    assert(size > 0);

    chk = hipMalloc(&data, size);
    if (chk != hipSuccess)
        return NULL;

    return data;
}

void *abcdk_cuda_alloc_z(size_t size)
{
    void *data;
    
    assert(size > 0);

    data = abcdk_cuda_alloc(size);
    if(!data)
        return NULL;

    abcdk_cuda_memset(data, 0, size);

    return data;
}

template <typename T>
ABCDK_INVOKE_GLOBAL void _abcdk_cuda_memset_2d2d(T *data, T value, size_t size)
{
    size_t tid = abcdk::cuda::grid::get_tid(2, 2);

    if (tid >= size)
        return;

    data[tid] = value;
}

void *abcdk_cuda_memset(void *dst, int val, size_t size)
{
    uint3 dim[2];

    /*2D-2D*/
    abcdk::cuda::grid::make_dim_dim(dim, size, 64);

    _abcdk_cuda_memset_2d2d<uint8_t><<<dim[0], dim[1]>>>((uint8_t *)dst, (uint8_t)val, size);

    return dst;
}

int abcdk_cuda_memcpy(void *dst, int dst_in_host, const void *src, int src_in_host, size_t size)
{
    hipMemcpyKind kind = hipMemcpyDefault;
    hipError_t chk;

    assert(dst != NULL && src != NULL && size > 0);

    if (src_in_host && dst_in_host)
        kind = hipMemcpyHostToHost;
    else if (src_in_host)
        kind = hipMemcpyHostToDevice;
    else if (dst_in_host)
        kind = hipMemcpyDeviceToHost;
    else
        kind = hipMemcpyDeviceToDevice;

    chk = hipMemcpy(dst, src, size, kind);

    if (chk != hipSuccess)
        return -1;

    return 0;
}

int abcdk_cuda_memcpy_2d(void *dst, size_t dst_pitch, size_t dst_x_bytes, size_t dst_y, int dst_in_host,
                         const void *src, size_t src_pitch, size_t src_x_bytes, size_t src_y, int src_in_host,
                         size_t roi_width_bytes, size_t roi_height)
{
    hip_Memcpy2D copy_args = {0};
    hipError_t chk;

    assert(dst != NULL && src != NULL && roi_width_bytes > 0 && roi_height > 0);

    copy_args.dstXInBytes = dst_x_bytes;
    copy_args.dstY = dst_y;
    copy_args.dstMemoryType = (dst_in_host ? hipMemoryTypeHost : hipMemoryTypeDevice);
    copy_args.dstHost = (dst_in_host ? dst : NULL);
    copy_args.dstDevice = (hipDeviceptr_t)(dst_in_host ? NULL : dst);
    copy_args.dstPitch = dst_pitch;

    copy_args.srcXInBytes = src_x_bytes;
    copy_args.srcY = src_y;
    copy_args.srcMemoryType = (src_in_host ? hipMemoryTypeHost : hipMemoryTypeDevice);
    copy_args.srcHost = (src_in_host ? src : NULL);
    copy_args.srcDevice = (hipDeviceptr_t)(src_in_host ? NULL : src);
    copy_args.srcPitch = src_pitch;

    copy_args.WidthInBytes = roi_width_bytes;
    copy_args.Height = roi_height;

    chk = hipMemcpyParam2D(&copy_args);

    if (chk != hipSuccess)
        return -1;

    return 0;
}

void *abcdk_cuda_copyfrom(const void *src, size_t size, int src_in_host)
{
    void *dst;
    int chk;

    assert(src != NULL && size > 0);

    dst = abcdk_cuda_alloc(size);
    if (!dst)
        return NULL;

    chk = abcdk_cuda_memcpy(dst, 0, src, src_in_host, size);
    if (chk == 0)
        return dst;

    abcdk_cuda_free(&dst);
    return NULL;
}

#endif //__cuda_cuda_h__
