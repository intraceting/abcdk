#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/imgproc.h"
#include "grid.cu.hxx"
#include "util.cu.hxx"

#ifdef __cuda_cuda_h__

template <typename T>
ABCDK_CUDA_GLOBAL void _abcdk_cuda_imgproc_brightness_2d2d(int channels, bool packed,
                                                           T *dst, size_t dst_ws, T *src, size_t src_ws,
                                                           size_t w, size_t h, float *alpha, float *bate)
{

    size_t tid = abcdk::cuda::grid_get_tid(2, 2);

    size_t y = tid / w;
    size_t x = tid % w;

    if (x >= w || y >= h)
        return;

    for (size_t z = 0; z < channels; z++)
    {
        size_t src_offset = abcdk::cuda::off<T>(packed, w, src_ws, h, channels, 0, x, y, z);
        size_t dst_offset = abcdk::cuda::off<T>(packed, w, dst_ws, h, channels, 0, x, y, z);

        dst[dst_offset] = (T)abcdk::cuda::pixel_clamp<float>(src[src_offset] * alpha[z] + bate[z]);
    }
}

template <typename T>
ABCDK_CUDA_HOST int _abcdk_cuda_imgproc_brightness(int channels, bool packed,
                                                   T *dst, size_t dst_ws, T *src, size_t src_ws,
                                                   size_t w, size_t h, float *alpha, float *bate)
{
    void *gpu_alpha = NULL, *gpu_bate = NULL;
    uint3 dim[2];

    gpu_alpha = abcdk_cuda_copyfrom(alpha, channels * sizeof(float), 1);
    gpu_bate = abcdk_cuda_copyfrom(bate, channels * sizeof(float), 1);

    if (!gpu_alpha || !gpu_bate)
    {
        abcdk_cuda_free(&gpu_alpha);
        abcdk_cuda_free(&gpu_bate);
        return -1;
    }

    /*2D-2D*/
    abcdk::cuda::grid_make_2d2d(dim, w * h, 64);

    _abcdk_cuda_imgproc_brightness_2d2d<T><<<dim[0], dim[1]>>>(channels, packed, dst, dst_ws, src, src_ws, w, h, (float *)gpu_alpha, (float *)gpu_bate);

    abcdk_cuda_free(&gpu_alpha);
    abcdk_cuda_free(&gpu_bate);
    return 0;
}

int abcdk_cuda_imgproc_brightness_8u_c1r(uint8_t *dst, size_t dst_ws, uint8_t *src, size_t src_ws,
                                         size_t w, size_t h, float *alpha, float *bate)
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(alpha != NULL && bate != NULL);

    return _abcdk_cuda_imgproc_brightness(1, true, dst, dst_ws, src, src_ws, w, h, alpha, bate);
}

int abcdk_cuda_imgproc_brightness_8u_c3r(uint8_t *dst, size_t dst_ws, uint8_t *src, size_t src_ws,
                                         size_t w, size_t h, float *alpha, float *bate)
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(alpha != NULL && bate != NULL);

    return _abcdk_cuda_imgproc_brightness(3, true, dst, dst_ws, src, src_ws, w, h, alpha, bate);
}

int abcdk_cuda_imgproc_brightness_8u_c4r(uint8_t *dst, size_t dst_ws, uint8_t *src, size_t src_ws,
                                         size_t w, size_t h, float *alpha, float *bate)
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(alpha != NULL && bate != NULL);

    return _abcdk_cuda_imgproc_brightness(4, true, dst, dst_ws, src, src_ws, w, h, alpha, bate);
}

#endif // __cuda_cuda_h__