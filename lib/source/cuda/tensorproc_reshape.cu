#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/tensorproc.h"
#include "../impl/tensorproc.hxx"
#include "grid.cu.hxx"

#ifdef __cuda_cuda_h__

template <typename T>
ABCDK_INVOKE_GLOBAL void _abcdk_cuda_tensorproc_reshape_2d2d(bool dst_packed, T *dst, size_t dst_b, size_t dst_w, size_t dst_ws, size_t dst_h, size_t dst_c,
                                                             bool src_packed, T *src, size_t src_b, size_t src_w, size_t src_ws, size_t src_h, size_t src_c)
{
    size_t tid = abcdk::cuda::grid::get_tid(2, 2);

    abcdk::tensorproc::reshape_kernel<T>(dst_packed, dst, dst_b, dst_w, dst_ws, dst_h, dst_c, src_packed, src, src_b, src_w, src_ws, src_h, src_c, tid);
}

template <typename T>
ABCDK_INVOKE_HOST int _abcdk_cuda_tensorproc_reshape(bool dst_packed, T *dst, size_t dst_b, size_t dst_w, size_t dst_ws, size_t dst_h, size_t dst_c,
                                                     bool src_packed, T *src, size_t src_b, size_t src_w, size_t src_ws, size_t src_h, size_t src_c)
{
    size_t dst_total, src_total;
    uint3 dim[2];

    assert(dst != NULL && dst_b > 0 && dst_w > 0 && dst_ws > 0 && dst_h > 0 && dst_c > 0);
    assert(dst != NULL && src_b > 0 && src_w > 0 && src_ws > 0 && src_h > 0 && src_c > 0);

    assert(dst_packed ? (dst_ws >= dst_w * dst_c * size_t(T)) : (dst_ws >= dst_w * size_t(T)));
    assert(src_packed ? (src_ws >= src_w * src_c * size_t(T)) : (src_ws >= src_w * size_t(T)));

    dst_total = dst_b * dst_w * dst_h * dst_c;
    src_total = src_b * src_w * src_h * src_c;

    assert(dst_total == src_total);

    /*2D-2D*/
    abcdk::cuda::grid::make_dim_dim(dim, dst_total, 64);

    _abcdk_cuda_tensorproc_reshape_2d2d<T><<<dim[0], dim[1]>>>(dst_packed, dst, dst_b, dst_w, dst_ws, dst_h, dst_c, src_packed, src, src_b, src_w, src_ws, src_h, src_c);

    return 0;
}

int abcdk_cuda_tensorproc_reshape_8u_R(bool dst_packed, uint8_t *dst, size_t dst_b, size_t dst_w, size_t dst_ws, size_t dst_h, size_t dst_c,
                                       bool src_packed, uint8_t *src, size_t src_b, size_t src_w, size_t src_ws, size_t src_h, size_t src_c)
{
    return _abcdk_cuda_tensorproc_reshape<uint8_t>(dst_packed, dst, dst_b, dst_w, dst_ws, dst_h, dst_c, src_packed, src, src_b, src_w, src_ws, src_h, src_c);
}

#endif // __cuda_cuda_h__
