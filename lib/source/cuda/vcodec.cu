/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/vcodec.h"
#include "vcodec_decoder_ffnv.cu.hxx"
#include "vcodec_decoder_aarch64.cu.hxx"
#include "vcodec_encoder_ffnv.cu.hxx"
#include "vcodec_encoder_aarch64.cu.hxx"

#ifdef __cuda_cuda_h__

/** CUDA视频编/解码器。*/
typedef struct _abcdk_cuda_vcodec
{
    /**是否为编码器。!0 是，0 否。*/
    uint8_t encoder;

    /**编码器。*/
    abcdk::cuda::vcodec::encoder *encoder_ctx;

    /**解码器。*/
    abcdk::cuda::vcodec::decoder *decoder_ctx;

} abcdk_cuda_vcodec_t;

static void _abcdk_cuda_vcodec_private_free_cb(void **ctx)
{
    abcdk_cuda_vcodec_t *ctx_p;

    if (!ctx || !*ctx)
        return;

    ctx_p = (abcdk_cuda_vcodec_t *)*ctx;
    *ctx = NULL;

    if (ctx_p->encoder)
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        abcdk::cuda::vcodec::encoder_ffnv::destory(&ctx_p->encoder_ctx);
#elif defined(__aarch64__)
        abcdk::cuda::vcodec::encoder_aarch64::destory(&ctx_p->encoder_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__
    }
    else
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        abcdk::cuda::vcodec::decoder_ffnv::destory(&ctx_p->decoder_ctx);
#elif defined(__aarch64__)
        abcdk::cuda::vcodec::decoder_aarch64::destory(&ctx_p->decoder_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__
    }

    abcdk_heap_free(ctx_p);
}

abcdk_media_vcodec_t *abcdk_cuda_vcodec_alloc(int encoder,hipCtx_t cuda_ctx)
{
    abcdk_media_vcodec_t *ctx;
    abcdk_cuda_vcodec_t *ctx_p;

    assert(cuda_ctx != NULL);

    ctx = abcdk_media_vcodec_alloc(ABCDK_MEDIA_TAG_CUDA);
    if (!ctx)
        return NULL;

    ctx->private_ctx_free_cb = _abcdk_cuda_vcodec_private_free_cb;

    /*创建内部对象。*/
    ctx->private_ctx = abcdk_heap_alloc(sizeof(abcdk_cuda_vcodec_t));
    if(!ctx->private_ctx)
        goto ERR;

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;
    
    if (ctx_p->encoder = encoder)
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        ctx_p->encoder_ctx = abcdk::cuda::vcodec::encoder_ffnv::create(cuda_ctx);
#elif defined(__aarch64__)
        ctx_p->encoder_ctx = abcdk::cuda::vcodec::encoder_aarch64::create(cuda_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__

        if (!ctx_p->encoder_ctx)
            goto ERR;
    }
    else
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        ctx_p->decoder_ctx = abcdk::cuda::vcodec::decoder_ffnv::create(cuda_ctx);
#elif defined(__aarch64__)
        ctx_p->decoder_ctx = abcdk::cuda::vcodec::decoder_aarch64::create(cuda_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__

        if (!ctx_p->decoder_ctx)
            goto ERR;
    }

    return ctx;

ERR:

    abcdk_media_vcodec_free(&ctx);

    return NULL;
}

int abcdk_cuda_vcodec_start(abcdk_media_vcodec_t *ctx, abcdk_media_vcodec_param_t *param)
{
    abcdk_cuda_vcodec_t *ctx_p;
    int chk;

    assert(ctx != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    if (ctx_p->encoder)
    {
        chk = ctx_p->encoder_ctx->open(param);
        if (chk != 0)
            return -1;
    }
    else
    {
        chk = ctx_p->decoder_ctx->open(param);
        if (chk != 0)
            return -1;
    }

    return 0;
}

int abcdk_cuda_vcodec_encode(abcdk_media_vcodec_t *ctx,abcdk_media_packet_t **dst, const abcdk_media_frame_t *src)
{
    abcdk_cuda_vcodec_t *ctx_p;

    assert(ctx != NULL && dst != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    ABCDK_ASSERT(ctx_p->encoder, "解码器不能用于编码。");

    return ctx_p->encoder_ctx->update(dst,src);
}

int abcdk_cuda_vcodec_decode(abcdk_media_vcodec_t *ctx,abcdk_media_frame_t **dst, const abcdk_media_packet_t *src)
{
    abcdk_cuda_vcodec_t *ctx_p;

    assert(ctx != NULL && dst != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    ABCDK_ASSERT(!ctx_p->encoder, "编码器不能用于解码。");

    return ctx_p->decoder_ctx->update(dst, src);
}

#else //__cuda_cuda_h__

void abcdk_cuda_vcodec_destroy(abcdk_media_vcodec_t **ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return ;
}

abcdk_media_vcodec_t *abcdk_cuda_vcodec_create(int encode, hipCtx_t cuda_ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return NULL;
}

int abcdk_cuda_vcodec_start(abcdk_media_vcodec_t *ctx, abcdk_media_vcodec_param_t *param)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_vcodec_encode(abcdk_media_vcodec_t *ctx,abcdk_media_packet_t **dst, const abcdk_media_frame_t *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_vcodec_decode(abcdk_media_vcodec_t *ctx,abcdk_media_frame_t **dst, const abcdk_media_packet_t *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

#endif //__cuda_cuda_h__
