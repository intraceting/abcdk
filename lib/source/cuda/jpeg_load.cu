/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/jpeg.h"

#ifdef __cuda_cuda_h__
#ifdef AVUTIL_AVUTIL_H


AVFrame *abcdk_cuda_jpeg_load(const char *src, hipCtx_t cuda_ctx)
{
    AVFrame *dst;
    abcdk_cuda_jpeg_t *ctx;

    assert(src != NULL && cuda_ctx != NULL);

    ctx = abcdk_cuda_jpeg_create(0,NULL,cuda_ctx);
    if(!ctx)
        return NULL;

    dst = abcdk_cuda_jpeg_decode_from_file(ctx,src);
    abcdk_cuda_jpeg_destroy(&ctx);
    
    return dst;
}


#endif // AVUTIL_AVUTIL_H
#endif //__cuda_cuda_h__
