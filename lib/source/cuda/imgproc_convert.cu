#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/cuda/imgproc.h"
#include "grid.cu.hxx"
#include "util.cu.hxx"

#ifdef __cuda_cuda_h__

template <class ST, class DT>
ABCDK_CUDA_GLOBAL void _abcdk_cuda_imgproc_convert_2d2d(int channels,
                                                        bool dst_packed, DT *dst, size_t dst_ws,
                                                        bool src_packed, ST *src, size_t src_ws,
                                                        size_t w, size_t h,
                                                        float *scale, float *mean, float *std)
{
    size_t tid = abcdk::cuda::grid_get_tid(2, 2);

    size_t y = tid / w;
    size_t x = tid % w;

    if (x >= w || y >= h)
        return;

    for (size_t z = 0; z < channels; z++)
    {
        size_t src_of = abcdk::cuda::off<ST>(src_packed, w, src_ws, h, channels, 0, x, y, z);
        size_t dst_of = abcdk::cuda::off<DT>(dst_packed, w, dst_ws, h, channels, 0, x, y, z);

        ST *src_p = abcdk::cuda::ptr<ST>(src, src_of);
        DT *dst_p = abcdk::cuda::ptr<DT>(dst, dst_of);
        *dst_p = (((DT)*src_p / scale[z]) - mean[z]) / std[z];
    }
}

template <class ST, class DT>
ABCDK_CUDA_HOST int _abcdk_cuda_imgproc_convert(int channels,
                                                bool dst_packed, DT *dst, size_t dst_ws,
                                                bool src_packed, ST *src, size_t src_ws,
                                                size_t w, size_t h,
                                                float *scale, float *mean, float *std)
{
    void *gpu_scale = NULL, *gpu_mean = NULL, *gpu_std = NULL;
    uint3 dim[2];

    gpu_scale = abcdk_cuda_copyfrom(scale, channels * sizeof(float), 1);
    gpu_mean = abcdk_cuda_copyfrom(mean, channels * sizeof(float), 1);
    gpu_std = abcdk_cuda_copyfrom(std, channels * sizeof(float), 1);

    if (!gpu_scale || !gpu_mean || !gpu_std)
    {
        abcdk_cuda_free(&gpu_scale);
        abcdk_cuda_free(&gpu_mean);
        abcdk_cuda_free(&gpu_std);
        return -1;
    }

    /*2D-2D*/
    abcdk::cuda::grid_make_2d2d(dim, w * h, 64);

    _abcdk_cuda_imgproc_convert_2d2d<ST, DT><<<dim[0], dim[1]>>>(channels, dst_packed, dst, dst_ws, src_packed, src, src_ws, w, h, (float *)gpu_scale, (float *)gpu_mean, (float *)gpu_std);

    abcdk_cuda_free(&gpu_scale);
    abcdk_cuda_free(&gpu_mean);
    abcdk_cuda_free(&gpu_std);
    return 0;
}

int abcdk_cuda_imgproc_convert_8u_to_32f_3r(int dst_packed, float *dst, size_t dst_ws,
                                            int src_packed, uint8_t *src, size_t src_ws,
                                            size_t w, size_t h, float scale[3], float mean[3], float std[3])
{
    assert(dst != NULL && dst_ws > 0);
    assert(src != NULL && src_ws > 0);
    assert(w > 0 && h > 0);
    assert(scale != NULL && mean != NULL && std != NULL);

    return _abcdk_cuda_imgproc_convert<uint8_t, float>(3, dst_packed, dst, dst_ws, src_packed, src, src_ws, w, h, scale, mean, std);
}

#endif // __cuda_cuda_h__
