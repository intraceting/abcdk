#include "hip/hip_runtime.h"
/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/nvidia/imgproc.h"
#include "../generic/imageproc.hxx"
#include "grid.cu.hxx"

#ifdef __cuda_cuda_h__

template <typename T>
ABCDK_INVOKE_GLOBAL void _abcdk_cuda_imgproc_drawrect_2d2d(int channels, bool packed,
                                                           T *dst, size_t w, size_t ws, size_t h,
                                                           T *color, int weight, int *corner)
{
    size_t tid = abcdk::cuda::grid::get_tid(2, 2);

    abcdk::generic::imageproc::drawrect<T>(channels, packed, dst, w, ws, h, color, weight, corner, tid);
}

template <typename T>
ABCDK_INVOKE_HOST int _abcdk_cuda_imgproc_drawrect(int channels, bool packed,
                                                   T *dst, size_t w, size_t ws, size_t h,
                                                   T *color, int weight, int *corner)
{
    void *gpu_color = NULL, *gpu_conrer = NULL;
    uint3 dim[2];

    assert(dst != NULL && w > 0 && ws > 0 && h > 0);
    assert(color != NULL && weight > 0 && corner != NULL);

    gpu_color = abcdk_cuda_copyfrom(color, channels * sizeof(T), 1);
    gpu_conrer = abcdk_cuda_copyfrom(corner, 4 * sizeof(int), 1);

    if (!gpu_color || !gpu_conrer)
    {
        abcdk_cuda_free(&gpu_color);
        abcdk_cuda_free(&gpu_conrer);
        return -1;
    }

    /*2D-2D*/
    abcdk::cuda::grid::make_dim_dim(dim, w * h, 64);

    _abcdk_cuda_imgproc_drawrect_2d2d<T><<<dim[0], dim[1]>>>(channels, packed, dst, w, ws, h, (T *)gpu_color, weight, (int *)gpu_conrer);
    abcdk_cuda_free(&gpu_color);
    abcdk_cuda_free(&gpu_conrer);

    return 0;
}

__BEGIN_DECLS

int abcdk_cuda_imgproc_drawrect_8u(abcdk_torch_image_t *dst, uint8_t color[], int weight, int corner[4])
{
    int dst_depth;

    assert(dst != NULL && color != NULL && weight > 0 && corner != NULL);
    assert(dst->pixfmt == ABCDK_TORCH_PIXFMT_GRAY8 ||
           dst->pixfmt == ABCDK_TORCH_PIXFMT_RGB24 ||
           dst->pixfmt == ABCDK_TORCH_PIXFMT_BGR24 ||
           dst->pixfmt == ABCDK_TORCH_PIXFMT_RGB32 ||
           dst->pixfmt == ABCDK_TORCH_PIXFMT_BGR32);

    dst_depth = abcdk_torch_pixfmt_channels(dst->pixfmt);

    return _abcdk_cuda_imgproc_drawrect<uint8_t>(dst_depth, true, dst->data[0], dst->width, dst->stride[0], dst->height, color, weight, corner);
}

__END_DECLS

#else // __cuda_cuda_h__

__BEGIN_DECLS

int abcdk_cuda_imgproc_drawrect_8u(abcdk_torch_image_t *dst, uint8_t color[], int weight, int corner[4])
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

__END_DECLS

#endif // __cuda_cuda_h__