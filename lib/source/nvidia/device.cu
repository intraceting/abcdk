/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/nvidia/device.h"


#ifdef __cuda_cuda_h__

int abcdk_cuda_get_device()
{
    int device = -1;
    hipError_t chk;

    chk = hipGetDevice(&device);
    if(chk != hipSuccess)
        return -1;

    return device;
}

int abcdk_cuda_set_device(int device)
{
    hipError_t chk;

    assert(device >=0);

    chk = hipSetDevice(device);
    if(chk != hipSuccess)
        return -1;

    return 0;
}

int abcdk_cuda_get_device_name(char name[256], int device)
{
    struct hipDeviceProp_t prop;
    hipError_t chk;

    assert(name != NULL && device >= 0);

    chk = hipGetDeviceProperties(&prop, device);
    if (chk != hipSuccess)
        return -1;

    strncpy(name, prop.name, 256);

    return 0;
}

int abcdk_cuda_get_runtime_version(int *minor)
{
    int num_ver = 0;
    int major = -1;
    hipError_t chk;

    chk = hipRuntimeGetVersion(&num_ver);
    if (chk != hipSuccess)
        return -1;
    
    major = num_ver / 1000;

    if (minor)
        *minor = (num_ver % 1000) / 10;

    return major;
}

void abcdk_cuda_ctx_destroy(hipCtx_t *ctx)
{
    hipCtx_t ctx_p;

    if(!ctx || !*ctx)
        return;

    ctx_p = *ctx;
    *ctx = NULL;

    hipCtxDestroy(ctx_p);
}

hipCtx_t abcdk_cuda_ctx_create(int device, int flag)
{
    hipCtx_t ctx;
    hipDevice_t cuda_dev;
    hipError_t chk;

    chk = hipDeviceGet(&cuda_dev, device);
    if (chk != hipSuccess)
        return NULL;

    chk = hipCtxCreate(&ctx, flag, cuda_dev);
    if (chk != hipSuccess)
        return NULL;

    return ctx;
}

int abcdk_cuda_ctx_push_current(hipCtx_t ctx)
{
    hipError_t chk;

    assert(ctx != NULL);

    chk = hipCtxPushCurrent(ctx);
    if (chk != hipSuccess)
        return -1;

    return 0;
}

int abcdk_cuda_ctx_pop_current(hipCtx_t *ctx)
{
    hipError_t chk;

    assert(ctx != NULL);

    chk = hipCtxPopCurrent(ctx);
    if (chk != hipSuccess)
        return -1;

    return 0;
}

#else //__cuda_cuda_h__

int abcdk_cuda_get_device()
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_set_device(int device)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_get_device_name(char name[256], int device)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_get_runtime_version(int *minor)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

void abcdk_cuda_ctx_destroy(hipCtx_t *ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
}

hipCtx_t abcdk_cuda_ctx_create(int device, int flag)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return NULL;
}

int abcdk_cuda_ctx_push_current(hipCtx_t ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_ctx_pop_current(hipCtx_t *ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

#endif //__cuda_cuda_h__