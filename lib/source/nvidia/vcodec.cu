/*
 * This file is part of ABCDK.
 *
 * Copyright (c) 2025 The ABCDK project authors. All Rights Reserved.
 *
 */
#include "abcdk/nvidia/vcodec.h"
#include "vcodec_decoder_ffnv.cu.hxx"
#include "vcodec_decoder_aarch64.cu.hxx"
#include "vcodec_encoder_ffnv.cu.hxx"
#include "vcodec_encoder_aarch64.cu.hxx"


__BEGIN_DECLS

#ifdef __cuda_cuda_h__

/** CUDA视频编/解码器。*/
typedef struct _abcdk_cuda_vcodec
{
    /**是否为编码器。!0 是，0 否。*/
    uint8_t encoder;

    /**编码器。*/
    abcdk::cuda::vcodec::encoder *encoder_ctx;

    /**解码器。*/
    abcdk::cuda::vcodec::decoder *decoder_ctx;

} abcdk_cuda_vcodec_t;

static void _abcdk_cuda_vcodec_private_free_cb(void **ctx)
{
    abcdk_cuda_vcodec_t *ctx_p;

    if (!ctx || !*ctx)
        return;

    ctx_p = (abcdk_cuda_vcodec_t *)*ctx;
    *ctx = NULL;

    if (ctx_p->encoder)
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        abcdk::cuda::vcodec::encoder_ffnv::destory(&ctx_p->encoder_ctx);
#elif defined(__aarch64__)
        abcdk::cuda::vcodec::encoder_aarch64::destory(&ctx_p->encoder_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__
    }
    else
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        abcdk::cuda::vcodec::decoder_ffnv::destory(&ctx_p->decoder_ctx);
#elif defined(__aarch64__)
        abcdk::cuda::vcodec::decoder_aarch64::destory(&ctx_p->decoder_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__
    }

    abcdk_heap_free(ctx_p);
}

abcdk_torch_vcodec_t *abcdk_cuda_vcodec_alloc(int encoder,hipCtx_t cuda_ctx)
{
    abcdk_torch_vcodec_t *ctx;
    abcdk_cuda_vcodec_t *ctx_p;

    assert(cuda_ctx != NULL);

    ctx = abcdk_torch_vcodec_alloc(ABCDK_TORCH_TAG_CUDA);
    if (!ctx)
        return NULL;

    ctx->private_ctx_free_cb = _abcdk_cuda_vcodec_private_free_cb;

    /*创建内部对象。*/
    ctx->private_ctx = abcdk_heap_alloc(sizeof(abcdk_cuda_vcodec_t));
    if(!ctx->private_ctx)
        goto ERR;

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;
    
    if (ctx_p->encoder = encoder)
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        ctx_p->encoder_ctx = abcdk::cuda::vcodec::encoder_ffnv::create(cuda_ctx);
#elif defined(__aarch64__)
        ctx_p->encoder_ctx = abcdk::cuda::vcodec::encoder_aarch64::create(cuda_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__

        if (!ctx_p->encoder_ctx)
            goto ERR;
    }
    else
    {
#ifdef FFNV_CUDA_DYNLINK_LOADER_H
        ctx_p->decoder_ctx = abcdk::cuda::vcodec::decoder_ffnv::create(cuda_ctx);
#elif defined(__aarch64__)
        ctx_p->decoder_ctx = abcdk::cuda::vcodec::decoder_aarch64::create(cuda_ctx);
#endif //FFNV_CUDA_DYNLINK_LOADER_H || __aarch64__

        if (!ctx_p->decoder_ctx)
            goto ERR;
    }

    return ctx;

ERR:

    abcdk_torch_vcodec_free(&ctx);

    return NULL;
}

int abcdk_cuda_vcodec_start(abcdk_torch_vcodec_t *ctx, abcdk_torch_vcodec_param_t *param)
{
    abcdk_cuda_vcodec_t *ctx_p;
    int chk;

    assert(ctx != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    if (ctx_p->encoder)
    {
        chk = ctx_p->encoder_ctx->open(param);
        if (chk != 0)
            return -1;
    }
    else
    {
        chk = ctx_p->decoder_ctx->open(param);
        if (chk != 0)
            return -1;
    }

    return 0;
}

int abcdk_cuda_vcodec_encode(abcdk_torch_vcodec_t *ctx,abcdk_torch_packet_t **dst, const abcdk_torch_frame_t *src)
{
    abcdk_cuda_vcodec_t *ctx_p;

    assert(ctx != NULL && dst != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    ABCDK_ASSERT(ctx_p->encoder, "解码器不能用于编码。");

    return ctx_p->encoder_ctx->update(dst,src);
}

int abcdk_cuda_vcodec_decode(abcdk_torch_vcodec_t *ctx,abcdk_torch_frame_t **dst, const abcdk_torch_packet_t *src)
{
    abcdk_cuda_vcodec_t *ctx_p;

    assert(ctx != NULL && dst != NULL);

    ctx_p = (abcdk_cuda_vcodec_t *)ctx->private_ctx;

    ABCDK_ASSERT(!ctx_p->encoder, "编码器不能用于解码。");

    return ctx_p->decoder_ctx->update(dst, src);
}

#ifdef AVCODEC_AVCODEC_H

int abcdk_cuda_vcodec_encode_to_ffmpeg(abcdk_torch_vcodec_t *ctx, AVPacket **dst, const abcdk_torch_frame_t *src)
{
    abcdk_torch_packet_t *tmp_dst = NULL;
    AVPacket *dst_p;
    int chk;

    assert(ctx != NULL && dst != NULL);

    chk = abcdk_cuda_vcodec_encode(ctx, &tmp_dst, src);
    if (chk > 0)
    {
        dst_p = *dst;

        if (dst_p)
            av_packet_unref(dst_p);
        else
            dst_p = *dst = av_packet_alloc();

        if (!dst_p)
        {
            abcdk_torch_packet_free(&tmp_dst);
            return -1;
        }

        av_new_packet(dst_p, tmp_dst->size);
        memcpy(dst_p->data, tmp_dst->data, tmp_dst->size);
    }

    abcdk_torch_packet_free(&tmp_dst);

    return chk;
}

int abcdk_cuda_vcodec_decode_from_ffmpeg(abcdk_torch_vcodec_t *ctx, abcdk_torch_frame_t **dst, const AVPacket *src)
{
    abcdk_torch_packet_t tmp_src = {0};

    assert(ctx != NULL && dst != NULL);

    if (src)
    {
        tmp_src.data = src->data;
        tmp_src.size = src->size;
        tmp_src.pts = src->pts;
    }

    return abcdk_cuda_vcodec_decode(ctx, dst, (src ? &tmp_src : NULL));
}

#endif //AVCODEC_AVCODEC_H

#else //__cuda_cuda_h__

void abcdk_cuda_vcodec_destroy(abcdk_torch_vcodec_t **ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return ;
}

abcdk_torch_vcodec_t *abcdk_cuda_vcodec_create(int encode, hipCtx_t cuda_ctx)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return NULL;
}

int abcdk_cuda_vcodec_start(abcdk_torch_vcodec_t *ctx, abcdk_torch_vcodec_param_t *param)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_vcodec_encode(abcdk_torch_vcodec_t *ctx,abcdk_torch_packet_t **dst, const abcdk_torch_frame_t *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_vcodec_decode(abcdk_torch_vcodec_t *ctx,abcdk_torch_frame_t **dst, const abcdk_torch_packet_t *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

#ifdef AVCODEC_AVCODEC_H

int abcdk_cuda_vcodec_encode_to_ffmpeg(abcdk_torch_vcodec_t *ctx, AVPacket **dst, const abcdk_torch_frame_t *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

int abcdk_cuda_vcodec_decode_from_ffmpeg(abcdk_torch_vcodec_t *ctx, abcdk_torch_frame_t **dst, const AVPacket *src)
{
    abcdk_trace_printf(LOG_WARNING, "当前环境在构建时未包含CUDA工具。");
    return -1;
}

#endif //AVCODEC_AVCODEC_H

#endif //__cuda_cuda_h__


__END_DECLS
